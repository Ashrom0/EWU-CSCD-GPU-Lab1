
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

typedef unsigned long long bignum;
//return 1 if it is a prime else return 0
//save as main.cu 
// CUDA kernel. Each thread takes care of one element of c
__host__ __device__ bignum checkIfValIsPrime(bignum number)
{
    if(number ==1) return (bignum) 0;	
    if (number == 2) return (bignum) 0;
    if (number % 2 == 0) return (bignum) 0;
    for (long divisor = 3; divisor < (number / 2); divisor += 2)
    {
        if (number % divisor == 0)
        {
            return (bignum) 0;
        }
    }
    return (bignum) 1;
}

__global__ void isPrime(double *a, bignum length)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<length){
	  a[id] = checkIfValIsPrime((bignum) id);
	}
}


int main( int argc, char* argv[] )
{
   if(argc < 2)
   {
       printf("Usage: prime upbound\n");
       exit(-1);
   }
   bignum N = (bignum) atoi(argv[1]);
   bignum blockSize  = (bignum) atoi(argv[2]);
   if(N <= 0)
   {
       printf("Usage: prime upbound, you input invalid upbound number!\n");
       exit(-1);
   }
 
    // Host input
    double *h_a;
    // Host output
    double *h_c;
    
    // Device input
    double *d_a;
    // Device output
    double *d_c;
 
    // Size, in bytes, of each vector
    size_t bytes = N*sizeof(double);

    // Allocate memory for vector on host
    h_a = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_c, bytes);
    printf("Made it past the allocation of memory\n"); 
    int i;
    // Initialize array with 0's to show that it is empty

    printf("Initialize array with 0's\n"); 
    for( i = 0; i < N; i++ ) {
        h_a[i] = 0;
    }
 
    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
	
    //Number of threads blocks in grid.  
    int gridSize = (int)ceil((float)(N+1)/2/blockSize);

    // Execute the kernel
    isPrime<<<gridSize, blockSize>>>(d_a, N);
 
    // Copy array back to host
    hipMemcpy( h_a, d_a, bytes, hipMemcpyDeviceToHost );
 
    // Sum up vector c and print result divided by n, this should equal 1 without error
    double sum = 0;
    printf("In the for block adding up sum\n");

    for(i=0; i<N; i++){
        sum += h_a[i];
	printf("In position %d ", i);
	printf("We have %f\n", h_a[i]);
    }
    printf("Final result: %f\n", sum);
 
    // Release device memory
    hipFree(d_a);
    hipFree(d_c);
 
    // Release host memory
    free(h_a);
    free(h_c);
 
    return 0;
}
